#include "hip/hip_runtime.h"
// my_gpu_math_library.cu
// Реализация операций с длинными векторами на видеокарте NVIDIA.
// 30.07.2021


#pragma once
#ifndef MY_GPU_MATH_LIBRARY_CU
#define MY_GPU_MATH_LIBRARY_CU 1

//#include "hip/hip_runtime.h"
//#include ""

//#include <stdio.h>

/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
*/


// false - используется центральный процессор.
 // Вычисление произведения матрицы на вектор происходит на видеокарте.
// Каждый раз данные передаются на видеокарту и освобождаются.
// Работает при включенном  MY_GPU_MATH_LIBRARY_CU_ON
#define MY_GPU_MATH_LIBRARY_CU_REALLOC_ON false
// Данные на видеокарту передаются минимальное число раз фактически единожды. 
// Во время потребности в вычислении произведения матрицы на вектор память видеокарты постоянно занята.
#define MY_GPU_MATH_LIBRARY_CU_ON  false 
#define EllFormat  true

// true если девайс ещё не инициализирован.
bool init_b_first_device = true;

#if MY_GPU_MATH_LIBRARY_CU_ON

hipError_t cudaStatus;

__global__ void initKernel(doublereal* x, integer n, doublereal x0)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i < n) {
		x[i] = x0;
		i += blockDim.x * gridDim.x;
	}

}
#endif

// bdevice == true запуск на GPU.
// инициализирует вектор x значением x0.
void init_v(doublereal*& x, integer n, doublereal x0, bool bdevice)
{

	if ((MY_GPU_MATH_LIBRARY_CU_ON) && (bdevice)) {

#if MY_GPU_MATH_LIBRARY_CU_ON	

		if (init_b_first_device) {

			

			int device;
			device = idevice_Tesla;

			// Choose which GPU to run on, change this on a multi-GPU system.
			cudaStatus = hipSetDevice(device);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
				system("PAUSE");
				exit(1);
			}

			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, device);
			printf("%s\n", prop.name);
			// GeForce 840M 384потока 1ГГц каждый март 2014 года. 28нм.

			init_b_first_device = false;
		}

		// на видеокарте GPU.

		doublereal* dev_x = nullptr;

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_x, (n) * sizeof(doublereal));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_x failed!");
			system("PAUSE");
			exit(1);
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_x, x, n * sizeof(doublereal), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_x, x HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		initKernel<<<128, 128 >>>(dev_x,n,x0);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "initKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			system("PAUSE");
			exit(1);
		}

		cudaStatus = hipMemcpy(x, dev_x, n * sizeof(doublereal), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy x, dev_x DeviceToHost failed!");
			system("PAUSE");
			exit(1);
		}

		hipFree(dev_x);

#endif

	}
	else {
		// на процессоре CPU.
#pragma omp parallel for
		for (integer i = 0; i < n; ++i) {
			x[i] = x0;
		}
	}
}

#if MY_GPU_MATH_LIBRARY_CU_ON

// реализация неэффективна как пишут в интернете
// Скалярная реализация.
__global__ void MatrixCRSByVectorKernelScalar(const doublereal* val, const integer* col_ind, const integer* row_ptr, const doublereal* V, doublereal* tmp, const integer n)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i < n) {
		//tmp[i] = 0.0;

		doublereal sum=0.0;
		const integer rowend = row_ptr[i + 1];
		const integer rowbeg = row_ptr[i];


		for (integer j = rowbeg; j < rowend; ++j)
		{
			sum += val[j] * V[col_ind[j]];
		}
		tmp[i] = sum;


		i += blockDim.x * gridDim.x;
	}

}

#define FULL_WARP_MASK 0xFFFFFFFF

template <class T>
__device__ T warp_reduce(T val)
{

	for (integer offset = warpSize / 2; offset > 0; offset /= 2)
	{
		val += __shfl_down_sync(FULL_WARP_MASK, val, offset);
	}

	return val;
}


// Векторная реализация.
__global__ void MatrixCRSByVectorKernel(const doublereal* val, const integer* col_ind, const integer* row_ptr, const doublereal* V, doublereal* tmp, const integer n)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int warp_id = i / 32;
	int lane = i % 32;

	int row = warp_id; //// One warp per raw

	

	while (row < n) {
		//tmp[i] = 0.0;

		doublereal sum = 0.0;

		const integer rowend = row_ptr[row + 1]; 
		const integer rowbeg = row_ptr[row];

		
		for (integer j = rowbeg + lane; j < rowend; j += 32)
		{
			sum += val[j] * V[col_ind[j]];
		}

		sum = warp_reduce(sum);

		if (lane == 0 && row < n) {
			tmp[row] = sum;
		}	


		i += blockDim.x * gridDim.x;
		warp_id = i / 32;
		lane = i % 32;

		row = warp_id; //// One warp per raw

	}

}


// Ellpack Itpack
__global__ void MatrixEllPackItpackByVectorKernel(const doublereal* data, const integer* indices, const integer string_size, const doublereal* V, doublereal* tmp, const integer n, const integer iadd)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i < n) {
		//tmp[i] = 0.0;

		doublereal sum = 0.0;
		
		for (integer j = 0; j < string_size; ++j)
		{
			const integer element_offset = i + j * n;
			const doublereal val = data[element_offset];
			//if (val != 0)
			sum +=  val * V[indices[element_offset]];
		}
		tmp[i+ iadd] = sum;


		i += blockDim.x * gridDim.x;
	}

}
#endif

void MatrixCRSByVector(doublereal*& val, integer*& col_ind, integer*& row_ptr, doublereal*& V, doublereal*& tmp, const integer n)
{

	if (MY_GPU_MATH_LIBRARY_CU_ON) {

#if MY_GPU_MATH_LIBRARY_CU_ON

		if (init_b_first_device) {

			int device;
			device = idevice_Tesla;

			// Choose which GPU to run on, change this on a multi-GPU system.
			cudaStatus = hipSetDevice(device);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
				system("PAUSE");
				exit(1);
			}

			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, device);
			printf("%s\n", prop.name);
			// GeForce 840M 384потока 1ГГц каждый март 2014 года. 28нм.

			init_b_first_device = false;
		}


#if EllFormat
		doublereal* data = nullptr;
		integer* indices = nullptr;
		integer string_size = -1;

		

		for (integer i77 = 0; i77 < n; ++i77) {
			if (row_ptr[i77 + 1] - row_ptr[i77] > string_size) {
				string_size = row_ptr[i77 + 1] - row_ptr[i77];
			}
			//if (i77>= maxelm) {
				//std::cout << "\nstart hvost=" << i77 << "  maxelm=" << maxelm << "start "<< row_ptr75[i77]  << "end=" << row_ptr75[i77 + 1] << std::endl;
				//system("pause");
			//}
		}

		data = new doublereal[string_size * n];
		indices = new integer[string_size * n];

		


		// Перепаковка в Ell формат.
#pragma omp parallel for
		for (integer i77 = 0; i77 < string_size * n; ++i77) {
			data[i77] = 0.0;
		}



#pragma omp parallel for
		for (integer i77 = 0; i77 < n; ++i77) {

			const integer rowend = row_ptr[i77 + 1];
			const integer rowbeg = row_ptr[i77];

			integer jstart = 0;
			for (integer j77 = rowbeg; j77 < rowend; ++j77)
			{
				const integer element_offset = i77 + jstart * n;
				data[element_offset] = val[j77];
				indices[element_offset] = col_ind[j77];

				++jstart;
			}
			for (integer j77 = jstart; j77 < string_size; ++j77)
			{
				const integer element_offset = i77 + j77 * n;

				if (indices[element_offset - 1] < n - 2) {
					indices[element_offset] = indices[element_offset - 1] + 1;
					data[element_offset] = 0.0;
				}
				else {
					indices[element_offset] = indices[element_offset - 1];
					data[element_offset] = 0.0;
				}

			}
		}

		doublereal* dev_data = nullptr;
		integer* dev_indices = nullptr;	

		doublereal* dev_V = nullptr;
		doublereal* dev_tmp = nullptr;

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_data, (n * string_size) * sizeof(doublereal));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_val failed!");
			system("PAUSE");
			exit(1);
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_indices, (n * string_size) * sizeof(integer));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_col_ind failed!");
			system("PAUSE");
			exit(1);
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_V, (n) * sizeof(doublereal));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_V failed!");
			system("PAUSE");
			exit(1);
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_tmp, (n) * sizeof(doublereal));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_tmp failed!");
			system("PAUSE");
			exit(1);
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_data, data, n * string_size * sizeof(doublereal), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_val, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_indices, indices, n * string_size * sizeof(integer), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_col_ind, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		delete[] data;
		data = nullptr;
		delete[] indices;
		indices = nullptr;

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_V, V, n * sizeof(doublereal), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_V, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		MatrixEllPackItpackByVectorKernel<<<128, 128>>>(dev_data, dev_indices, string_size, dev_V, dev_tmp, n, 0);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, " MatrixEllPackItpackByVectorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			system("PAUSE");
			exit(1);
		}

		cudaStatus = hipMemcpy(tmp, dev_tmp, n * sizeof(doublereal), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy tmp, dev_tmp DeviceToHost failed!");
			system("PAUSE");
			exit(1);
		}


		hipFree(dev_data);
		hipFree(dev_indices);

		hipFree(dev_V);
		hipFree(dev_tmp);

#else

		doublereal* dev_val = nullptr;
		integer* dev_col_ind = nullptr;
		integer* dev_row_ptr = nullptr;
		doublereal* dev_V = nullptr;
		doublereal* dev_tmp = nullptr;

		integer nnz = row_ptr[n];

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_val, (nnz) * sizeof(doublereal));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_val failed!");
			system("PAUSE");
			exit(1);
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_col_ind, (nnz) * sizeof(integer));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_col_ind failed!");
			system("PAUSE");
			exit(1);
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_row_ptr, (n+1) * sizeof(integer));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_row_ptr failed!");
			system("PAUSE");
			exit(1);
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_V, (n) * sizeof(doublereal));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_V failed!");
			system("PAUSE");
			exit(1);
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_tmp, (n) * sizeof(doublereal));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_tmp failed!");
			system("PAUSE");
			exit(1);
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_val, val, nnz * sizeof(doublereal), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_val, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_col_ind, col_ind, nnz * sizeof(integer), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_col_ind, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}


		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_row_ptr, row_ptr, (n+1) * sizeof(integer), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_rho_ptr, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_V, V, n * sizeof(doublereal), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_V, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_tmp, tmp, n * sizeof(doublereal), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_tmp, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		MatrixCRSByVectorKernel<<<128,128>>>(dev_val, dev_col_ind, dev_row_ptr,dev_V, dev_tmp,n);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MatrixCRSByVectorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			system("PAUSE");
			exit(1);
		}

		cudaStatus = hipMemcpy(tmp, dev_tmp, n * sizeof(doublereal), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy tmp, dev_tmp DeviceToHost failed!");
			system("PAUSE");
			exit(1);
		}

		hipFree(dev_tmp);
		hipFree(dev_val);
		hipFree(dev_col_ind);
		hipFree(dev_V);
		hipFree(dev_row_ptr);

#endif

#endif

	}
	else {
		// на процессоре CPU.

		if (number_cores() == 1) {

			// Если у нас только один поток то лучше вообще не писать #pragma omp parallel for
			// т.к. с #pragma omp parallel for будет медленней

			for (integer i = 0; i < n; ++i) {
				doublereal sum = 0.0;
				const integer rowend = row_ptr[i + 1];
				const integer rowbeg = row_ptr[i];
				
				for (integer j = rowbeg; j < rowend; ++j)
				{
					sum += val[j] * V[col_ind[j]];
				}
				tmp[i] = sum;
			}
		}
		else {

			// вектор tmp индексируется начиная с нуля так же как и вектор V
	//#pragma omp parallel for
		//	for (integer i = 0; i < n; ++i) tmp[i] = 0.0;

			// В целях увеличения быстродействия
			// вся необходимая память выделяется заранее.
			//if (tmp == nullptr)
			//{
			//printf("malloc: out of memory for vector tmp in MatrixCRSByVector\n"); // нехватка памяти
			//system("pause");
			//exit(0);  // завершение программы
			//}



			//omp_set_num_threads(inumcore);

#pragma omp parallel for  schedule (guided)
			for (integer i = 0; i < n; ++i) {
				doublereal sum = 0.0;
				const integer rowend = row_ptr[i + 1];
				const integer rowbeg = row_ptr[i];

				
				for (integer j = rowbeg; j < rowend; ++j)
				{
					sum += val[j] * V[col_ind[j]];
				}
				tmp[i] = sum;
			}

			//return tmp;
		}
	}
} // MatrixCRSByVector

#endif