#include "hip/hip_runtime.h"
// my_gpu_math_library.cu
// Реализация операций с длинными векторами на видеокарте.
// 30.07.2021


#pragma once
#ifndef MY_GPU_MATH_LIBRARY_CU
#define MY_GPU_MATH_LIBRARY_CU 1

// false - используется центральный процессор.
#define MY_GPU_MATH_LIBRARY_CU_ON  false 
#define EllFormat  false

// true если девайс ещё не инициализирован.
bool init_b_first_device = true;

#if MY_GPU_MATH_LIBRARY_CU_ON

hipError_t cudaStatus;

__global__ void initKernel(doublereal* x, integer n, doublereal x0)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i < n) {
		x[i] = x0;
		i += blockDim.x * gridDim.x;
	}

}
#endif

// bdevice == true запуск на GPU.
// инициализирует вектор x значением x0.
void init_v(doublereal*& x, integer n, doublereal x0, bool bdevice)
{

	if ((MY_GPU_MATH_LIBRARY_CU_ON)&&(bdevice)) {

#if MY_GPU_MATH_LIBRARY_CU_ON	

		if (init_b_first_device) {

			

			int device;
			device = idevice_Tesla;

			// Choose which GPU to run on, change this on a multi-GPU system.
			cudaStatus = hipSetDevice(device);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
				system("PAUSE");
				exit(1);
			}

			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, device);
			printf("%s\n", prop.name);
			// GeForce 840M 384потока 1ГГц каждый март 2014 года. 28нм.

			init_b_first_device = false;
		}

		// на видеокарте GPU.

		doublereal* dev_x = nullptr;

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_x, (n) * sizeof(doublereal));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_x failed!");
			system("PAUSE");
			exit(1);
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_x, x, n * sizeof(doublereal), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_x, x HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		initKernel<<<128, 128 >>>(dev_x,n,x0);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "initKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			system("PAUSE");
			exit(1);
		}

		cudaStatus = hipMemcpy(x, dev_x, n * sizeof(doublereal), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy x, dev_x DeviceToHost failed!");
			system("PAUSE");
			exit(1);
		}

		hipFree(dev_x);

#endif

	}
	else {
		// на процессоре CPU.
#pragma omp parallel for
		for (integer i = 0; i < n; i++) {
			x[i] = x0;
		}
	}
}

#if MY_GPU_MATH_LIBRARY_CU_ON

// реализация неэффективна как пишут в интернете
// Скалярная реализация.
__global__ void MatrixCRSByVectorKernelScalar(const doublereal* val, const integer* col_ind, const integer* row_ptr, const doublereal* V, doublereal* tmp, const integer n)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i < n) {
		//tmp[i] = 0.0;

		doublereal sum=0.0;
		const integer rowend = row_ptr[i + 1];
		const integer rowbeg = row_ptr[i];


		for (integer j = rowbeg; j < rowend; ++j)
		{
			sum += val[j] * V[col_ind[j]];
		}
		tmp[i] = sum;


		i += blockDim.x * gridDim.x;
	}

}

#define FULL_WARP_MASK 0xFFFFFFFF

template <class T>
__device__ T warp_reduce(T val)
{

	for (integer offset = warpSize / 2; offset > 0; offset /= 2)
	{
		val += __shfl_down_sync(FULL_WARP_MASK, val, offset);
	}

	return val;
}


// Векторная реализация.
__global__ void MatrixCRSByVectorKernel(const doublereal* val, const integer* col_ind, const integer* row_ptr, const doublereal* V, doublereal* tmp, const integer n)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int warp_id = i / 32;
	int lane = i % 32;

	int row = warp_id; //// One warp per raw

	

	while (row < n) {
		//tmp[i] = 0.0;

		doublereal sum = 0.0;

		const integer rowend = row_ptr[row + 1]; 
		const integer rowbeg = row_ptr[row];

		
		for (integer j = rowbeg + lane; j < rowend; j += 32)
		{
			sum += val[j] * V[col_ind[j]];
		}

		sum = warp_reduce(sum);

		if (lane == 0 && row < n) {
			tmp[row] = sum;
		}	


		i += blockDim.x * gridDim.x;
		warp_id = i / 32;
		lane = i % 32;

		row = warp_id; //// One warp per raw

	}

}


// Ellpack Itpack
__global__ void MatrixEllPackItpackByVectorKernel(const doublereal* data, const integer* indices, const integer string_size, const doublereal* V, doublereal* tmp, const integer n, const integer iadd)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i < n) {
		//tmp[i] = 0.0;

		doublereal sum = 0.0;
		
		for (integer j = 0; j < string_size; ++j)
		{
			const integer element_offset = i + j * n;
			const doublereal val = data[element_offset];
			//if (val != 0)
			sum +=  val * V[indices[element_offset]];
		}
		tmp[i+ iadd] = sum;


		i += blockDim.x * gridDim.x;
	}

}
#endif

void MatrixCRSByVector(doublereal*& val, integer*& col_ind, integer*& row_ptr, doublereal*& V, doublereal*& tmp, const integer n)
{

	if (MY_GPU_MATH_LIBRARY_CU_ON) {

#if MY_GPU_MATH_LIBRARY_CU_ON

		if (init_b_first_device) {

			int device;
			device = idevice_Tesla;

			// Choose which GPU to run on, change this on a multi-GPU system.
			cudaStatus = hipSetDevice(device);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
				system("PAUSE");
				exit(1);
			}

			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, device);
			printf("%s\n", prop.name);
			// GeForce 840M 384потока 1ГГц каждый март 2014 года. 28нм.

			init_b_first_device = false;
		}


		doublereal* dev_val = nullptr;
		integer* dev_col_ind = nullptr;
		integer* dev_row_ptr = nullptr;
		doublereal* dev_V = nullptr;
		doublereal* dev_tmp = nullptr;

		integer nnz = row_ptr[n];

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_val, (nnz) * sizeof(doublereal));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_val failed!");
			system("PAUSE");
			exit(1);
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_col_ind, (nnz) * sizeof(integer));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_col_ind failed!");
			system("PAUSE");
			exit(1);
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_row_ptr, (n+1) * sizeof(integer));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_row_ptr failed!");
			system("PAUSE");
			exit(1);
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_V, (n) * sizeof(doublereal));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_V failed!");
			system("PAUSE");
			exit(1);
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_tmp, (n) * sizeof(doublereal));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc dev_tmp failed!");
			system("PAUSE");
			exit(1);
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_val, val, nnz * sizeof(doublereal), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_val, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_col_ind, col_ind, nnz * sizeof(integer), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_col_ind, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}


		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_row_ptr, row_ptr, (n+1) * sizeof(integer), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_rho_ptr, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_V, V, n * sizeof(doublereal), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_V, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_tmp, tmp, n * sizeof(doublereal), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy dev_tmp, val HostToDevice failed!");
			system("PAUSE");
			exit(1);
		}

		MatrixCRSByVectorKernel<<<128,128>>>(dev_val, dev_col_ind, dev_row_ptr,dev_V, dev_tmp,n);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MatrixCRSByVectorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			system("PAUSE");
			exit(1);
		}

		cudaStatus = hipMemcpy(tmp, dev_tmp, n * sizeof(doublereal), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy tmp, dev_tmp DeviceToHost failed!");
			system("PAUSE");
			exit(1);
		}

		hipFree(dev_tmp);
		hipFree(dev_val);
		hipFree(dev_col_ind);
		hipFree(dev_V);
		hipFree(dev_row_ptr);

#endif

	}
	else {
		// на процессоре CPU.

		if (number_cores() == 1) {

			// Если у нас только один поток то лучше вообще не писать #pragma omp parallel for
			// т.к. с #pragma omp parallel for будет медленней

			for (integer i = 0; i < n; ++i) {
				doublereal sum = 0.0;
				const integer rowend = row_ptr[i + 1];
				const integer rowbeg = row_ptr[i];
				
				for (integer j = rowbeg; j < rowend; ++j)
				{
					sum += val[j] * V[col_ind[j]];
				}
				tmp[i] = sum;
			}
		}
		else {

			// вектор tmp индексируется начиная с нуля так же как и вектор V
	//#pragma omp parallel for
		//	for (integer i = 0; i < n; ++i) tmp[i] = 0.0;

			// В целях увеличения быстродействия
			// вся необходимая память выделяется заранее.
			//if (tmp == nullptr)
			//{
			//printf("malloc: out of memory for vector tmp in MatrixCRSByVector\n"); // нехватка памяти
			//getchar();
			//exit(0);  // завершение программы
			//}



			//omp_set_num_threads(inumcore);

#pragma omp parallel for  schedule (guided)
			for (integer i = 0; i < n; ++i) {
				doublereal sum = 0.0;
				const integer rowend = row_ptr[i + 1];
				const integer rowbeg = row_ptr[i];

				
				for (integer j = rowbeg; j < rowend; ++j)
				{
					sum += val[j] * V[col_ind[j]];
				}
				tmp[i] = sum;
			}

			//return tmp;
		}
	}
} // MatrixCRSByVector

#endif